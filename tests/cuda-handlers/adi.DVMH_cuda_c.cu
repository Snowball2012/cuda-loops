#include "hip/hip_runtime.h"
#include <cassert>

/* DVMH includes */
#include <dvmhlib2.h>
#include <dvmh_cuda_helpers.h>
#include <hip/hip_runtime.h>

__global__ void loop_adi_41_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3) {
    /* Parameters */
    DvmhArrayHelper3<double> a(a_base, a_hdr1, a_hdr2);
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
            for (k = boundsLow_3; k <= boundsHigh_3; k++)
            {
                a[i][j][k] = (a[i - 1][j][k] + a[i + 1][j][k]) / 2;
            }
}




extern "C" void loop_adi_41_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_41_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_41_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_41_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], boundsLow[2], boundsHigh[2]);
        restBlocks -= blocks.x;
    }
}

__global__ void loop_adi_46_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3) {
    /* Parameters */
    DvmhArrayHelper3<double> a(a_base, a_hdr1, a_hdr2);
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
            for (k = boundsLow_3; k <= boundsHigh_3; k++)
            {
                a[i][j][k] = (a[i][j - 1][k] + a[i][j + 1][k]) / 2;
            }
}

extern "C" void loop_adi_46_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_46_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_46_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_46_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], boundsLow[2], boundsHigh[2]);
        restBlocks -= blocks.x;
    }
}

__global__ void loop_adi_51_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3, double eps, double eps_grid[]) {
    /* Parameters */
    DvmhArrayHelper3<double> a(a_base, a_hdr1, a_hdr2);
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
            for (k = boundsLow_3; k <= boundsHigh_3; k++)
            {
              double tmp1 = (a[i][j][k - 1] + a[i][j][k + 1]) / 2;
              double tmp2 = fabs(a[i][j][k] - tmp1);
              eps = ((eps) > (tmp2) ? (eps) : (tmp2));
              a[i][j][k] = tmp1;
            }

    /* Write reduction values to global memory */
    eps_grid[0] = eps;
}

extern "C" void loop_adi_51_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_51_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_51_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* Reductions-related stuff */
    double eps;
    double *eps_grid;
    dvmh_loop_cuda_register_red_C(loop_ref, 1, (void **)&eps_grid, 0);
    dvmh_loop_red_init_C(loop_ref, 1, &eps, 0);
    dvmh_loop_cuda_red_prepare_C(loop_ref, 1, 1, 0);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_51_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], boundsLow[2], boundsHigh[2], eps, eps_grid);
        restBlocks -= blocks.x;
    }

    dvmh_loop_cuda_red_finish_C(loop_ref, 1);
}

__global__ void loop_adi_92_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType blocksS_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3, CudaIndexType blocksS_3, CudaIndexType blockOffset) {
    /* Parameters */
    DvmhArrayHelper3<double> a(a_base, a_hdr1, a_hdr2);
    /* Supplementary variables for loop handling */
    CudaIndexType restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks / blocksS_2;
    i = boundsLow_1 + (curBlocks * blockDim.z + threadIdx.z);
    if (i <= boundsHigh_1) {
        restBlocks = restBlocks - curBlocks * blocksS_2;
        curBlocks = restBlocks / blocksS_3;
        j = boundsLow_2 + (curBlocks * blockDim.y + threadIdx.y);
        if (j <= boundsHigh_2) {
            restBlocks = restBlocks - curBlocks * blocksS_3;
            curBlocks = restBlocks;
            k = boundsLow_3 + (curBlocks * blockDim.x + threadIdx.x);
            if (k <= boundsHigh_3)
            {
                if (k == 0 || k == 384 - 1 || j == 0 || j == 384 - 1 || i == 0 || i == 384 - 1)
                  a[i][j][k] = 10. * i / (384 - 1) + 10. * j / (384 - 1) + 10. * k / (384 - 1);
                else
                  a[i][j][k] = 0;
            }
        }
    }
}

extern "C" void loop_adi_92_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType blocksS[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_92_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_92_cuda_kernel_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    blocksS[2] = ((boundsHigh[2] - boundsLow[2]) / loopSteps[2] + 1 + (threads.x - 1)) / threads.x;
    blocksS[1] = blocksS[2] * (((boundsHigh[1] - boundsLow[1]) / loopSteps[1] + 1 + (threads.y - 1)) / threads.y);
    blocksS[0] = blocksS[1] * (((boundsHigh[0] - boundsLow[0]) / loopSteps[0] + 1 + (threads.z - 1)) / threads.z);

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_92_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], blocksS[1], boundsLow[2], boundsHigh[2], blocksS[2], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}

