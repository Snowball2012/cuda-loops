#include "hip/hip_runtime.h"
#include <cassert>

/* DVMH includes */
#include <dvmhlib2.h>
#include <hiprand/hiprand_kernel.h>

int where_dep(int n, DvmType type_of_run, DvmType *idxs, int dep) {
    int count = 0;
    int h = 0;
    int hd = dep;
    for (int i = n - 1; i >= 0; --i) {
        if (type_of_run % 2 != 0) {
            count++;
            idxs[h] = i;
            h++;
        } else {
            idxs[hd] = i;
            hd++;
        }
        type_of_run = type_of_run / 2;
    }
    return count;
}




__global__ void loop_adi_41_cuda_kernel(double a_gen_0[], DvmType a_gen_0_hdr_gen_01, DvmType a_gen_0_hdr_gen_02, DvmType a_gen_0_hdr_gen_03, DvmType num_elem_x, DvmType base_i, DvmType loopSteps_2, DvmType idxs_2, DvmType num_elem_y, DvmType base_j, DvmType loopSteps_1, DvmType idxs_1, DvmType base_k, DvmType loopSteps_0, DvmType idxs_0)
{
    double* a_gen_0_base_gen_0 = a_gen_0;

    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    DvmType coords[3];
    DvmType id_x, id_y;
    DvmType red_idx1, red_idx2;

    id_x = blockIdx.x * blockDim.x + threadIdx.x;
    id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x < num_elem_x && id_y < num_elem_y) {
        coords[idxs_0] = base_i;
        coords[idxs_1] = base_j + id_x * loopSteps_2;
        coords[idxs_2] = basdeie_k + id_y * loopSteps_1;
        i = coords[0];
        j = coords[1];
        k = coords[2];

        {
                a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] = (a_gen_0_base_gen_0[(i - 1)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] + a_gen_0_base_gen_0[(i + 1)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)]) / 2;
            }
    }
}

extern "C" void loop_adi_41_cuda_across(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[], int dep_mask)
{
    DvmType tmpVar;

    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    dvmh_loop_autotransform_C(loop_ref, a_gen_0_hdr_gen_0);    double * a_gen_0 = (double *)
        dvmh_get_natural_base_C(device_num, a_gen_0_hdr_gen_0);
    DvmType a_gen_0_devHdr_gen_0[6];
    tmpVar = dvmh_fill_header_C(device_num, a_gen_0, a_gen_0_hdr_gen_0, a_gen_0_devHdr_gen_0, 0);
    assert(tmpVar == 0 || tmpVar == 1);

    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3], idxs[3];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_41_cuda_kernel_int_regs;
    DvmType shared_mem = 0;
    dvmh_loop_cuda_get_config_C(loop_ref, shared_mem, loop_adi_41_cuda_kernel_int_regs,
        &threads, &stream, &shared_mem);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    where_dep(3, dep_mask, idxs, 1);

    DvmType base_i = boundsLow[2];
    DvmType num_elem_x = (abs(boundsLow[1] - boundsHigh[1]) + 1) / abs(loopSteps[1]) + ((abs(boundsLow[1] - boundsHigh[1]) + 1) % abs(loopSteps[1]) != 0);
    blocks.x = num_elem_x / threads.x + ((num_elem_x % threads.x != 0)?1:0);
    DvmType base_j = boundsLow[1];
    DvmType num_elem_y = (abs(boundsLow[0] - boundsHigh[0]) + 1) / abs(loopSteps[0]) + ((abs(boundsLow[0] - boundsHigh[0]) + 1) % abs(loopSteps[0]) != 0);
    blocks.y = num_elem_y / threads.y + ((num_elem_y % threads.y != 0)?1:0);
    DvmType base_k = boundsLow[0];

    boundsHigh[2] = (abs(boundsHigh[2] - boundsLow[2]) + 1) / abs(loopSteps[2]);

    /* Reduction-related stuff */

    /* GPU execution */
    for (int tmpV = 0; tmpV < boundsHigh[2]; base_i += loopSteps[2], tmpV++) {
        loop_adi_41_cuda_kernel<<<blocks, threads, shared_mem, stream>>>(a_gen_0, a_gen_0_devHdr_gen_0[1], a_gen_0_devHdr_gen_0[2], a_gen_0_devHdr_gen_0[3], num_elem_x, base_i, loopSteps[2], idxs[2], num_elem_y, base_j, loopSteps[1], idxs[1], base_k, loopSteps[0], idxs[0]);
    }
}


extern "C" void loop_adi_41_cuda(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[])
{
    int dep_mask = dvmh_loop_get_dependency_mask_C(*pLoopRef);
    if (dep_mask == 1 || dep_mask == 2 || dep_mask == 4) {
        loop_adi_41_cuda_across(pLoopRef, a_gen_0_hdr_gen_0, dep_mask);
    }
}

__global__ void loop_adi_46_cuda_kernel(double a_gen_0[], DvmType a_gen_0_hdr_gen_01, DvmType a_gen_0_hdr_gen_02, DvmType a_gen_0_hdr_gen_03, DvmType num_elem_x, DvmType base_i, DvmType loopSteps_2, DvmType idxs_2, DvmType num_elem_y, DvmType base_j, DvmType loopSteps_1, DvmType idxs_1, DvmType base_k, DvmType loopSteps_0, DvmType idxs_0)
{
    double* a_gen_0_base_gen_0 = a_gen_0;

    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    DvmType coords[3];
    DvmType id_x, id_y;
    DvmType red_idx1, red_idx2;

    id_x = blockIdx.x * blockDim.x + threadIdx.x;
    id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x < num_elem_x && id_y < num_elem_y) {
        coords[idxs_0] = base_i;
        coords[idxs_1] = base_j + id_x * loopSteps_2;
        coords[idxs_2] = base_k + id_y * loopSteps_1;
        i = coords[0];
        j = coords[1];
        k = coords[2];

        {
                a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] = (a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j - 1)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] + a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j + 1)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)]) / 2;
            }
    }
}

extern "C" void loop_adi_46_cuda_across(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[], int dep_mask)
{
    DvmType tmpVar;

    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    dvmh_loop_autotransform_C(loop_ref, a_gen_0_hdr_gen_0);    double * a_gen_0 = (double *)
        dvmh_get_natural_base_C(device_num, a_gen_0_hdr_gen_0);
    DvmType a_gen_0_devHdr_gen_0[6];
    tmpVar = dvmh_fill_header_C(device_num, a_gen_0, a_gen_0_hdr_gen_0, a_gen_0_devHdr_gen_0, 0);
    assert(tmpVar == 0 || tmpVar == 1);

    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3], idxs[3];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_46_cuda_kernel_int_regs;
    DvmType shared_mem = 0;
    dvmh_loop_cuda_get_config_C(loop_ref, shared_mem, loop_adi_46_cuda_kernel_int_regs,
        &threads, &stream, &shared_mem);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    where_dep(3, dep_mask, idxs, 1);

    DvmType base_i = boundsLow[2];
    DvmType num_elem_x = (abs(boundsLow[1] - boundsHigh[1]) + 1) / abs(loopSteps[1]) + ((abs(boundsLow[1] - boundsHigh[1]) + 1) % abs(loopSteps[1]) != 0);
    blocks.x = num_elem_x / threads.x + ((num_elem_x % threads.x != 0)?1:0);
    DvmType base_j = boundsLow[1];
    DvmType num_elem_y = (abs(boundsLow[0] - boundsHigh[0]) + 1) / abs(loopSteps[0]) + ((abs(boundsLow[0] - boundsHigh[0]) + 1) % abs(loopSteps[0]) != 0);
    blocks.y = num_elem_y / threads.y + ((num_elem_y % threads.y != 0)?1:0);
    DvmType base_k = boundsLow[0];

    boundsHigh[2] = (abs(boundsHigh[2] - boundsLow[2]) + 1) / abs(loopSteps[2]);

    /* Reduction-related stuff */

    /* GPU execution */
    for (int tmpV = 0; tmpV < boundsHigh[2]; base_i += loopSteps[2], tmpV++) {
        loop_adi_46_cuda_kernel<<<blocks, threads, shared_mem, stream>>>(a_gen_0, a_gen_0_devHdr_gen_0[1], a_gen_0_devHdr_gen_0[2], a_gen_0_devHdr_gen_0[3], num_elem_x, base_i, loopSteps[2], idxs[2], num_elem_y, base_j, loopSteps[1], idxs[1], base_k, loopSteps[0], idxs[0]);
    }
}


extern "C" void loop_adi_46_cuda(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[])
{
    int dep_mask = dvmh_loop_get_dependency_mask_C(*pLoopRef);
    if (dep_mask == 1 || dep_mask == 2 || dep_mask == 4) {
        loop_adi_46_cuda_across(pLoopRef, a_gen_0_hdr_gen_0, dep_mask);
    }
}

__global__ void loop_adi_51_cuda_kernel(double a_gen_0[], DvmType a_gen_0_hdr_gen_01, DvmType a_gen_0_hdr_gen_02, DvmType a_gen_0_hdr_gen_03, DvmType num_elem_x, DvmType base_i, DvmType loopSteps_2, DvmType idxs_2, DvmType num_elem_y, DvmType base_j, DvmType loopSteps_1, DvmType idxs_1, DvmType base_k, DvmType loopSteps_0, DvmType idxs_0, double eps, double * eps_grid)
{
    double* a_gen_0_base_gen_0 = a_gen_0;

    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;
    extern __shared__ double eps_block[];

    DvmType coords[3];
    DvmType id_x, id_y;
    DvmType red_idx1, red_idx2;

    id_x = blockIdx.x * blockDim.x + threadIdx.x;
    id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x < num_elem_x && id_y < num_elem_y) {
        coords[idxs_0] = base_i;
        coords[idxs_1] = base_j + id_x * loopSteps_2;
        coords[idxs_2] = base_k + id_y * loopSteps_1;
        i = coords[0];
        j = coords[1];
        k = coords[2];

        {
              double tmp1 = (a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k - 1)*(a_gen_0_hdr_gen_03)] + a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k + 1)*(a_gen_0_hdr_gen_03)]) / 2;
              double tmp2 = fabs(a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] - tmp1);
              eps = ((eps) > (tmp2) ? (eps) : (tmp2));
              a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] = tmp1;
            }
    }

// Reduction for var eps
    id_x = blockDim.x * blockDim.y * blockDim.z / 2;
    red_idx1 = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y);
    eps_block[red_idx1] = eps;
    __syncthreads();
    red_idx2 = id_x;
    while (red_idx2 >= 1) {;
        __syncthreads();
        if (red_idx1 < red_idx2) {
            eps_block[red_idx1] = max(eps_block[red_idx1], eps_block[red_idx1 + red_idx2]);        }
            red_idx2 = red_idx2 / 2;
    }
    if (red_idx1 == 0) {
        eps_grid[blockIdx.x + (blockIdx.y + blockIdx.z * gridDim.y) * gridDim.x] = max(eps_grid[blockIdx.x + (blockIdx.y + blockIdx.z * gridDim.y) * gridDim.x], eps_block[0]);
    }
}

extern "C" void loop_adi_51_cuda_across(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[], int dep_mask)
{
    DvmType tmpVar;

    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    dvmh_loop_autotransform_C(loop_ref, a_gen_0_hdr_gen_0);    double * a_gen_0 = (double *)
        dvmh_get_natural_base_C(device_num, a_gen_0_hdr_gen_0);
    DvmType a_gen_0_devHdr_gen_0[6];
    tmpVar = dvmh_fill_header_C(device_num, a_gen_0, a_gen_0_hdr_gen_0, a_gen_0_devHdr_gen_0, 0);
    assert(tmpVar == 0 || tmpVar == 1);

    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3], idxs[3];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_51_cuda_kernel_int_regs;
    DvmType shared_mem = 16;
    dvmh_loop_cuda_get_config_C(loop_ref, shared_mem, loop_adi_51_cuda_kernel_int_regs,
        &threads, &stream, &shared_mem);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    where_dep(3, dep_mask, idxs, 1);

    DvmType base_i = boundsLow[2];
    DvmType num_elem_x = (abs(boundsLow[1] - boundsHigh[1]) + 1) / abs(loopSteps[1]) + ((abs(boundsLow[1] - boundsHigh[1]) + 1) % abs(loopSteps[1]) != 0);
    blocks.x = num_elem_x / threads.x + ((num_elem_x % threads.x != 0)?1:0);
    DvmType base_j = boundsLow[1];
    DvmType num_elem_y = (abs(boundsLow[0] - boundsHigh[0]) + 1) / abs(loopSteps[0]) + ((abs(boundsLow[0] - boundsHigh[0]) + 1) % abs(loopSteps[0]) != 0);
    blocks.y = num_elem_y / threads.y + ((num_elem_y % threads.y != 0)?1:0);
    DvmType base_k = boundsLow[0];

    boundsHigh[2] = (abs(boundsHigh[2] - boundsLow[2]) + 1) / abs(loopSteps[2]);

    /* Reduction-related stuff */
    double eps;
    double * eps_grid;
    dvmh_loop_cuda_register_red_C(loop_ref, 1, (void**)&eps_grid, 0);
    dvmh_loop_red_init_C(loop_ref, 1, &eps, 0);
    dvmh_loop_cuda_red_prepare_C(loop_ref, 1, blocks.x * blocks.y * blocks.z, 1);

    /* GPU execution */
    for (int tmpV = 0; tmpV < boundsHigh[2]; base_i += loopSteps[2], tmpV++) {
        loop_adi_51_cuda_kernel<<<blocks, threads, shared_mem, stream>>>(a_gen_0, a_gen_0_devHdr_gen_0[1], a_gen_0_devHdr_gen_0[2], a_gen_0_devHdr_gen_0[3], num_elem_x, base_i, loopSteps[2], idxs[2], num_elem_y, base_j, loopSteps[1], idxs[1], base_k, loopSteps[0], idxs[0], eps, eps_grid);
    }
    dvmh_loop_cuda_red_finish_C(loop_ref, 1);
}


extern "C" void loop_adi_51_cuda(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[])
{
    int dep_mask = dvmh_loop_get_dependency_mask_C(*pLoopRef);
    if (dep_mask == 1 || dep_mask == 2 || dep_mask == 4) {
        loop_adi_51_cuda_across(pLoopRef, a_gen_0_hdr_gen_0, dep_mask);
    }
}

__global__ void loop_adi_92_cuda_kernel(double a_gen_0[], DvmType a_gen_0_hdr_gen_01, DvmType a_gen_0_hdr_gen_02, DvmType a_gen_0_hdr_gen_03, DvmType boundsLow_1, DvmType boundsHigh_1, DvmType loopSteps_1, DvmType boundsLow_2, DvmType boundsHigh_2, DvmType loopSteps_2, DvmType blocksS_2, DvmType boundsLow_3, DvmType boundsHigh_3, DvmType loopSteps_3, DvmType blocksS_3, DvmType blockOffset)
{
    double* a_gen_0_base_gen_0 = a_gen_0;
    DvmType restBlocks, curBlocks;

    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks / blocksS_2;
    i = boundsLow_1 + (loopSteps_1) * ( curBlocks * blockDim.z + threadIdx.z);
    if (i <= boundsHigh_1)  {
        restBlocks = restBlocks - curBlocks * blocksS_2;
        curBlocks = restBlocks / blocksS_3;
        j = boundsLow_2 + (loopSteps_2) * ( curBlocks * blockDim.y + threadIdx.y);
        if (j <= boundsHigh_2)  {
            restBlocks = restBlocks - curBlocks * blocksS_3;
            curBlocks = restBlocks;
            k = boundsLow_3 + (loopSteps_3) * ( curBlocks * blockDim.x + threadIdx.x);
            if (k <= boundsHigh_3) 
            {
                if (k == 0 || k == 384 - 1 || j == 0 || j == 384 - 1 || i == 0 || i == 384 - 1)
                  a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] = 10. * i / (384 - 1) + 10. * j / (384 - 1) + 10. * k / (384 - 1);
                else
                  a_gen_0_base_gen_0[(i)*(a_gen_0_hdr_gen_01) + (j)*(a_gen_0_hdr_gen_02) + (k)*(a_gen_0_hdr_gen_03)] = 0;
            }
        }
    }
}

extern "C" void loop_adi_92_cuda(DvmType *pLoopRef, DvmType a_gen_0_hdr_gen_0[])
{
    DvmType tmpVar;

    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    dvmh_loop_autotransform_C(loop_ref, a_gen_0_hdr_gen_0);    double * a_gen_0 = (double *)
        dvmh_get_natural_base_C(device_num, a_gen_0_hdr_gen_0);
    DvmType a_gen_0_devHdr_gen_0[6];
    tmpVar = dvmh_fill_header_C(device_num, a_gen_0, a_gen_0_hdr_gen_0, a_gen_0_devHdr_gen_0, 0);
    assert(tmpVar == 0 || tmpVar == 1);

    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    DvmType blocksS[3];
    DvmType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_92_cuda_kernel_int_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_92_cuda_kernel_int_regs,
        &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[2] = ((boundsHigh[2] - boundsLow[2]) / loopSteps[2] + 1 + (threads.x - 1)) / threads.x;
    blocksS[1] = blocksS[2] * ((boundsHigh[1] - boundsLow[1]) / loopSteps[1] + 1 + (threads.y - 1)) / threads.y;
    blocksS[0] = blocksS[1] * ((boundsHigh[0] - boundsLow[0]) / loopSteps[0] + 1 + (threads.z - 1)) / threads.z;

    /* Reduction-related stuff */

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_92_cuda_kernel<<<blocks, threads, 0, stream>>>(a_gen_0, a_gen_0_devHdr_gen_0[1], a_gen_0_devHdr_gen_0[2], a_gen_0_devHdr_gen_0[3], boundsLow[0], boundsHigh[0], loopSteps[0], boundsLow[1], boundsHigh[1], loopSteps[1], blocksS[1], boundsLow[2], boundsHigh[2], loopSteps[2], blocksS[2], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}
