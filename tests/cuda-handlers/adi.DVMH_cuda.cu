#include "hip/hip_runtime.h"

#include <dvmhlib_cuda.h>
#define dcmplx2 Complex<double>
#define cmplx2 Complex<float>

      
      
      

//--------------------- Kernel for loop on line 16 ---------------------

         __global__ void   loop_adi_16_cuda_kernel_1_case(double a[], CudaIndexType a0003, CudaIndexType a0002, DvmType num_elem_j, DvmType num_elem_k, DvmType base_i, DvmType base_j, DvmType base_k, DvmType step_i, DvmType step_j, DvmType step_k, DvmType type_of_run, DvmType idxs_0, DvmType idxs_1, DvmType idxs_2){
            DvmType id_x, id_y;
            DvmType coords[3];

// Local needs
            int k, j, i;
            id_x = blockIdx.x * blockDim.x + threadIdx.x;
            id_y = blockIdx.y * blockDim.y + threadIdx.y;
            if (id_x < num_elem_j && id_y < num_elem_k) 
            {
               coords[idxs_0] = base_i;
               coords[idxs_1] = base_j + id_x * step_j;
               coords[idxs_2] = base_k + id_y * step_k;
               i = coords[0];
               j = coords[1];
               k = coords[2];

// Loop body
               a[i + a0003 * j + a0002 * k] = (a[i - 1 + a0003 * j + a0002 * k] + a[i + 1 + a0003 * j + a0002 * k]) / 2;
            }
         }


//--------------------- Kernel for loop on line 24 ---------------------

         __global__ void   loop_adi_24_cuda_kernel_1_case(double a[], CudaIndexType a0003, CudaIndexType a0002, DvmType num_elem_j, DvmType num_elem_k, DvmType base_i, DvmType base_j, DvmType base_k, DvmType step_i, DvmType step_j, DvmType step_k, DvmType type_of_run, DvmType idxs_0, DvmType idxs_1, DvmType idxs_2){
            DvmType id_x, id_y;
            DvmType coords[3];

// Local needs
            int k, j, i;
            id_x = blockIdx.x * blockDim.x + threadIdx.x;
            id_y = blockIdx.y * blockDim.y + threadIdx.y;
            if (id_x < num_elem_j && id_y < num_elem_k) 
            {
               coords[idxs_0] = base_i;
               coords[idxs_1] = base_j + id_x * step_j;
               coords[idxs_2] = base_k + id_y * step_k;
               i = coords[0];
               j = coords[1];
               k = coords[2];

// Loop body
               a[i + a0003 * j + a0002 * k] = (a[i + a0003 * (j - 1) + a0002 * k] + a[i + a0003 * (j + 1) + a0002 * k]) / 2;
            }
         }


//--------------------- Kernel for loop on line 33 ---------------------

         __global__ void   loop_adi_33_cuda_kernel_1_case(double a[], CudaIndexType a0003, CudaIndexType a0002, double eps, double eps_grid[], DvmType num_elem_j, DvmType num_elem_k, DvmType base_i, DvmType base_j, DvmType base_k, DvmType step_i, DvmType step_j, DvmType step_k, DvmType type_of_run, DvmType idxs_0, DvmType idxs_1, DvmType idxs_2){
            DvmType id_x, id_y;
            DvmType coords[3];

// Local needs
            int k, j, i;
            extern __shared__ double eps_block[];
            id_x = blockIdx.x * blockDim.x + threadIdx.x;
            id_y = blockIdx.y * blockDim.y + threadIdx.y;
            if (id_x < num_elem_j && id_y < num_elem_k) 
            {
               coords[idxs_0] = base_i;
               coords[idxs_1] = base_j + id_x * step_j;
               coords[idxs_2] = base_k + id_y * step_k;
               i = coords[0];
               j = coords[1];
               k = coords[2];

// Loop body
               eps = max(eps, fabs(a[i + a0003 * j + a0002 * k] - (a[i + a0003 * j + a0002 * (k - 1)] + a[i + a0003 * j + a0002 * (k + 1)]) / 2));
               a[i + a0003 * j + a0002 * k] = (a[i + a0003 * j + a0002 * (k - 1)] + a[i + a0003 * j + a0002 * (k + 1)]) / 2;
            }
            id_x = blockDim.x * blockDim.y * blockDim.z / 2;

// Reduction
            i = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y);
            eps_block[i] = eps;
            __syncthreads();
            j = id_x;
            while (j >= 1)
            {
               __syncthreads();
               if (i < j) 
               {
                  eps_block[i] = max(eps_block[i], eps_block[i + j]);
               }
               j = j / 2;
            }
            if (i == 0) 
            {
               eps_grid[blockIdx.x + (blockIdx.y + blockIdx.z * gridDim.y) * gridDim.x] = max(eps_grid[blockIdx.x + (blockIdx.y + blockIdx.z * gridDim.y) * gridDim.x], eps_block[0]);
            }
         }


//--------------------- Kernel for loop on line 74 ---------------------

         __global__ void   loop_adi_74_cuda_kernel(double a[], CudaIndexType a0003, CudaIndexType a0002, CudaIndexType blocks[], int nz, int ny, int nx){

// Local needs
            CudaIndexType k, j, i;
            int ibof;

// Calculate each thread's loop variables' values
            ibof = blockIdx.x * 6;
            k = blocks[ibof + 0] + threadIdx.z;
            if (k <= blocks[ibof + 1]) 
            {
               j = blocks[ibof + 2] + threadIdx.y;
               if (j <= blocks[ibof + 3]) 
               {
                  i = blocks[ibof + 4] + threadIdx.x;
                  if (i <= blocks[ibof + 5]) 
                  {

// Loop body
                     if (k == 1 | k == nz | j == 1 | j == ny | i == 1 | i == nx) 
                     {
                        a[i + a0003 * j + a0002 * k] = 10. * (i - 1) / (nx - 1) + 10. * (j - 1) / (ny - 1) + 10. * (k - 1) / (nz - 1);
                     }
                     else{
                        a[i + a0003 * j + a0002 * k] = 0.000000e+00;
                     }
                  }
               }
            }
         }

      

#ifdef _MS_F_
#define loop_adi_16_cuda_ loop_adi_16_cuda
#define loop_adi_24_cuda_ loop_adi_24_cuda
#define loop_adi_33_cuda_ loop_adi_33_cuda
#define loop_adi_74_cuda_ loop_adi_74_cuda
#endif
#include <cstdio>
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))

      extern "C" {
         extern  DvmType loop_adi_74_cuda_kernel_regs, loop_adi_33_1_case_regs, loop_adi_24_1_case_regs, loop_adi_16_1_case_regs;


//    CUDA handler for loop on line 16 

         void   loop_adi_16_cuda_1_case(DvmhLoopRef *loop_ref, DvmType a[], DvmType type_of_run){
            void   *a_base;
            DvmType d_a[6];
            hipStream_t stream;
            DvmType shared_mem;
            dim3 blocks, threads;
            int base_i, base_j, base_k;
            DvmType num_elem_y;
            DvmType num_elem_x;
            DvmType num_y;
            DvmType num_x;
            DvmType idxs[5];
            DvmType lowI[5], highI[5], idxI[5];
            DvmType device_num;

// Get device number
            device_num = loop_get_device_num_(loop_ref);

// Get natural bases
            a_base = dvmh_get_natural_base(&device_num, a);

// Fill device headers
            dvmh_fill_header_(&device_num, a_base, a, d_a);

// Get bounds
            loop_fill_bounds_(loop_ref, lowI, highI, idxI);

// Swap bounds
            dvmh_change_filled_bounds(lowI, highI, idxI, 3, 1, type_of_run, idxs);

// Get cuda config params
            threads = dim3(32, 8, 1);
            shared_mem = 0;
            loop_cuda_get_config(loop_ref, shared_mem, loop_adi_16_1_case_regs, &threads, &stream, &shared_mem);
            num_x = threads.x;
            num_y = threads.y;

//Start method
            blocks = dim3(1, 1, 1);
            base_i = lowI[2];
            base_j = lowI[1];
            num_elem_x = (abs(lowI[1] - highI[1]) + 1) / abs(idxI[1]) + ((abs(lowI[1] - highI[1]) + 1) % abs(idxI[1]) != 0);
            blocks.x = num_elem_x / num_x + (num_elem_x % num_x != 0);
            threads.x = num_x;
            base_k = lowI[0];
            num_elem_y = (abs(lowI[0] - highI[0]) + 1) / abs(idxI[0]) + ((abs(lowI[0] - highI[0]) + 1) % abs(idxI[0]) != 0);
            blocks.y = num_elem_y / num_y + (num_elem_y % num_y != 0);
            threads.y = num_y;
            highI[2] = (abs(highI[2] - lowI[2]) + 1) / abs(idxI[2]);
            for (int tmpV = 0 ; tmpV < highI[2] ; base_i = base_i + idxI[2], tmpV = tmpV + 1)
            {
               loop_adi_16_cuda_kernel_1_case<<<blocks, threads, shared_mem, stream>>>((double *)a_base, d_a[2], d_a[1], num_elem_x, num_elem_y, base_i, base_j, base_k, idxI[2], idxI[1], idxI[0], type_of_run, idxs[0], idxs[1], idxs[2]);
            }
         }


//    CUDA handler for loop on line 16 

         void   loop_adi_16_cuda_(DvmhLoopRef *loop_ref, DvmType a[]){
            int which_run;
            which_run = loop_get_dependency_mask_(loop_ref);
            if (which_run == 1 || which_run == 2 || which_run == 4) 
            {
               loop_adi_16_cuda_1_case(loop_ref, a, which_run);
            }
         }


//    CUDA handler for loop on line 24 

         void   loop_adi_24_cuda_1_case(DvmhLoopRef *loop_ref, DvmType a[], DvmType type_of_run){
            void   *a_base;
            DvmType d_a[6];
            hipStream_t stream;
            DvmType shared_mem;
            dim3 blocks, threads;
            int base_i, base_j, base_k;
            DvmType num_elem_y;
            DvmType num_elem_x;
            DvmType num_y;
            DvmType num_x;
            DvmType idxs[5];
            DvmType lowI[5], highI[5], idxI[5];
            DvmType device_num;

// Get device number
            device_num = loop_get_device_num_(loop_ref);

// Get natural bases
            a_base = dvmh_get_natural_base(&device_num, a);

// Fill device headers
            dvmh_fill_header_(&device_num, a_base, a, d_a);

// Get bounds
            loop_fill_bounds_(loop_ref, lowI, highI, idxI);

// Swap bounds
            dvmh_change_filled_bounds(lowI, highI, idxI, 3, 1, type_of_run, idxs);

// Get cuda config params
            threads = dim3(32, 8, 1);
            shared_mem = 0;
            loop_cuda_get_config(loop_ref, shared_mem, loop_adi_24_1_case_regs, &threads, &stream, &shared_mem);
            num_x = threads.x;
            num_y = threads.y;

//Start method
            blocks = dim3(1, 1, 1);
            base_i = lowI[2];
            base_j = lowI[1];
            num_elem_x = (abs(lowI[1] - highI[1]) + 1) / abs(idxI[1]) + ((abs(lowI[1] - highI[1]) + 1) % abs(idxI[1]) != 0);
            blocks.x = num_elem_x / num_x + (num_elem_x % num_x != 0);
            threads.x = num_x;
            base_k = lowI[0];
            num_elem_y = (abs(lowI[0] - highI[0]) + 1) / abs(idxI[0]) + ((abs(lowI[0] - highI[0]) + 1) % abs(idxI[0]) != 0);
            blocks.y = num_elem_y / num_y + (num_elem_y % num_y != 0);
            threads.y = num_y;
            highI[2] = (abs(highI[2] - lowI[2]) + 1) / abs(idxI[2]);
            for (int tmpV = 0 ; tmpV < highI[2] ; base_i = base_i + idxI[2], tmpV = tmpV + 1)
            {
               loop_adi_24_cuda_kernel_1_case<<<blocks, threads, shared_mem, stream>>>((double *)a_base, d_a[2], d_a[1], num_elem_x, num_elem_y, base_i, base_j, base_k, idxI[2], idxI[1], idxI[0], type_of_run, idxs[0], idxs[1], idxs[2]);
            }
         }


//    CUDA handler for loop on line 24 

         void   loop_adi_24_cuda_(DvmhLoopRef *loop_ref, DvmType a[]){
            int which_run;
            which_run = loop_get_dependency_mask_(loop_ref);
            if (which_run == 1 || which_run == 2 || which_run == 4) 
            {
               loop_adi_24_cuda_1_case(loop_ref, a, which_run);
            }
         }


//    CUDA handler for loop on line 33 

         void   loop_adi_33_cuda_1_case(DvmhLoopRef *loop_ref, DvmType a[], DvmType type_of_run){
            void   *a_base;
            DvmType d_a[6];
            hipStream_t stream;
            DvmType shared_mem;
            dim3 blocks, threads;
            int base_i, base_j, base_k;
            DvmType num_elem_y;
            DvmType num_elem_x;
            DvmType num_y;
            DvmType num_x;
            DvmType idxs[5];
            DvmType lowI[5], highI[5], idxI[5];
            DvmType num_of_red_blocks;
            double  *cuda_ptr_0;
            double eps;
            DvmType tmpVar1;
            DvmType tmpVar;
            DvmType device_num;

// Get device number
            device_num = loop_get_device_num_(loop_ref);

// Register reduction for CUDA-execution
            tmpVar = 1;
            loop_cuda_register_red(loop_ref, tmpVar, (void  **)&cuda_ptr_0, 0);
            loop_red_init_(loop_ref, &tmpVar, &eps, 0);

// Get natural bases
            a_base = dvmh_get_natural_base(&device_num, a);

// Fill device headers
            dvmh_fill_header_(&device_num, a_base, a, d_a);

// Get bounds
            loop_fill_bounds_(loop_ref, lowI, highI, idxI);

// Swap bounds
            dvmh_change_filled_bounds(lowI, highI, idxI, 3, 1, type_of_run, idxs);

// Get cuda config params
            threads = dim3(32, 8, 1);
            shared_mem = 8;
            loop_cuda_get_config(loop_ref, shared_mem, loop_adi_33_1_case_regs, &threads, &stream, &shared_mem);
            num_x = threads.x;
            num_y = threads.y;

//Start method
            blocks = dim3(1, 1, 1);
            base_i = lowI[2];
            base_j = lowI[1];
            num_elem_x = (abs(lowI[1] - highI[1]) + 1) / abs(idxI[1]) + ((abs(lowI[1] - highI[1]) + 1) % abs(idxI[1]) != 0);
            blocks.x = num_elem_x / num_x + (num_elem_x % num_x != 0);
            threads.x = num_x;
            base_k = lowI[0];
            num_elem_y = (abs(lowI[0] - highI[0]) + 1) / abs(idxI[0]) + ((abs(lowI[0] - highI[0]) + 1) % abs(idxI[0]) != 0);
            blocks.y = num_elem_y / num_y + (num_elem_y % num_y != 0);
            threads.y = num_y;
            num_of_red_blocks = blocks.x * blocks.y * blocks.z;
            tmpVar1 = 1;
            tmpVar = 1;
            loop_cuda_red_prepare(loop_ref, tmpVar, num_of_red_blocks, tmpVar1);
            highI[2] = (abs(highI[2] - lowI[2]) + 1) / abs(idxI[2]);
            for (int tmpV = 0 ; tmpV < highI[2] ; base_i = base_i + idxI[2], tmpV = tmpV + 1)
            {
               loop_adi_33_cuda_kernel_1_case<<<blocks, threads, shared_mem, stream>>>((double *)a_base, d_a[2], d_a[1], eps, cuda_ptr_0, num_elem_x, num_elem_y, base_i, base_j, base_k, idxI[2], idxI[1], idxI[0], type_of_run, idxs[0], idxs[1], idxs[2]);
            }
            tmpVar = 1;
            loop_red_finish(loop_ref, tmpVar);
         }


//    CUDA handler for loop on line 33 

         void   loop_adi_33_cuda_(DvmhLoopRef *loop_ref, DvmType a[]){
            int which_run;
            which_run = loop_get_dependency_mask_(loop_ref);
            if (which_run == 1 || which_run == 2 || which_run == 4) 
            {
               loop_adi_33_cuda_1_case(loop_ref, a, which_run);
            }
         }


//    CUDA handler for loop on line 74 

         void   loop_adi_74_cuda_(DvmhLoopRef *loop_ref, DvmType a[], int *nz, int *ny, int *nx){
            void   *a_base;
            DvmType d_a[6];
            dim3 blocks, threads;
            hipStream_t stream;
            CudaIndexType  *blocks_info;
            DvmType device_num;

// Get device number
            device_num = loop_get_device_num_(loop_ref);

// Get 'natural' bases
            a_base = dvmh_get_natural_base(&device_num, a);

// Fill 'device' headers
            dvmh_fill_header_(&device_num, a_base, a, d_a);

// Get CUDA configuration parameters
            threads = dim3(0, 0, 0);
            loop_cuda_get_config(loop_ref, 0, loop_adi_74_cuda_kernel_regs, &threads, &stream, 0);

// GPU execution
            while (loop_cuda_do(loop_ref, &blocks, &blocks_info) != 0)
            {
               loop_adi_74_cuda_kernel<<<blocks, threads, 0, stream>>>((double *)a_base, d_a[2], d_a[1], blocks_info, *nz, *ny, *nx);
            }
         }

      }
