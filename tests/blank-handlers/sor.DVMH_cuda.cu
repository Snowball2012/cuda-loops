#include "hip/hip_runtime.h"
#include <cassert>

/* DVMH includes */
#include <dvmhlib2.h>
#include <dvmh_cuda_helpers.h>

__global__ void loop_sor_28_cuda_kernel(float *A_base, CudaIndexType A_hdr1, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType blocksS_2, CudaIndexType blockOffset) {
    /* Parameters */
    DvmhArrayHelper2<float> A(A_base, A_hdr1);
    /* Supplementary variables for loop handling */
    CudaIndexType restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;
    int j;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks / blocksS_2;
    i = boundsLow_1 + (curBlocks * blockDim.y + threadIdx.y);
    if (i <= boundsHigh_1) {
        restBlocks = restBlocks - curBlocks * blocksS_2;
        curBlocks = restBlocks;
        j = boundsLow_2 + (curBlocks * blockDim.x + threadIdx.x);
        if (j <= boundsHigh_2)
        {
            if (i == j)
              A[i][j] = 10 + 2;
            else
              A[i][j] = -1.F;
        }
    }
}

extern "C" void loop_sor_28_cuda(DvmType *pLoopRef, DvmType A_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    float *A = (float *)dvmh_get_natural_base_C(device_num, A_hdr);
    DvmType A_devHdr[5];
    tmpVar = dvmh_fill_header_C(device_num, A, A_hdr, A_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[2], boundsHigh[2], loopSteps[2];
    CudaIndexType blocksS[2];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_sor_28_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_sor_28_cuda_kernel_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[1] = ((boundsHigh[1] - boundsLow[1]) / loopSteps[1] + 1 + (threads.x - 1)) / threads.x;
    blocksS[0] = blocksS[1] * (((boundsHigh[0] - boundsLow[0]) / loopSteps[0] + 1 + (threads.y - 1)) / threads.y);

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_sor_28_cuda_kernel<<<blocks, threads, 0, stream>>>(A, A_devHdr[1], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], blocksS[1], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}

__global__ void loop_sor_46_cuda_kernel(float *A_base, CudaIndexType A_hdr1, float *w_ptr, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, float eps, float eps_grid[]) {
    /* Parameters */
    DvmhArrayHelper2<float> A(A_base, A_hdr1);
    float &w = *w_ptr;
    /* User variables - loop index variables and other private variables */
    int i;
    int j;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
        {
          float s;
          s = A[i][j];
          A[i][j] = (w / 4) * (A[i - 1][j] + A[i + 1][j] + A[i][j - 1] + A[i][j + 1]) + (1 - w) * A[i][j];
          eps = ((fabs(s - A[i][j])) > (eps) ? (fabs(s - A[i][j])) : (eps));
        }

    /* Write reduction values to global memory */
    eps_grid[0] = eps;
}

extern "C" void loop_sor_46_cuda(DvmType *pLoopRef, DvmType A_hdr[], float *w_ptr) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    float *A = (float *)dvmh_get_natural_base_C(device_num, A_hdr);
    DvmType A_devHdr[5];
    tmpVar = dvmh_fill_header_C(device_num, A, A_hdr, A_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    float *w = (float *)dvmh_get_device_addr_C(device_num, w_ptr);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[2], boundsHigh[2], loopSteps[2];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_sor_46_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_sor_46_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* Reductions-related stuff */
    float eps;
    float *eps_grid;
    dvmh_loop_cuda_register_red_C(loop_ref, 1, (void **)&eps_grid, 0);
    dvmh_loop_red_init_C(loop_ref, 1, &eps, 0);
    dvmh_loop_cuda_red_prepare_C(loop_ref, 1, 1, 0);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_sor_46_cuda_kernel<<<blocks, threads, 0, stream>>>(A, A_devHdr[1], w, boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], eps, eps_grid);
        restBlocks -= blocks.x;
    }

    dvmh_loop_cuda_red_finish_C(loop_ref, 1);
}

