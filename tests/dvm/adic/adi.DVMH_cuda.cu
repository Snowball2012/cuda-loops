#include "hip/hip_runtime.h"
#include <cassert>

/* DVMH includes */
#include <dvmhlib2.h>
#include <dvmh_cuda_helpers.h>
#include <hiprand/hiprand_kernel.h>

int where_dep(int *ret, int n, DvmType type_of_run, DvmType *idxs, int dep) {
    int count = 0;
    int h = 0;
    int hd = dep;
    for (int i = 0; i < n; ++i) {
        ret[i] = 0;
        if (type_of_run % 2 != 0) {
            ret[i] = 1;
            count++;
            idxs[h] = i;
            h++;
        } else {
            idxs[hd] = i;
            hd++;
        }
        type_of_run = type_of_run / 2;
    }
    return count;
}

void change_filled_bounds(DvmType *low, DvmType *high, DvmType *idx, DvmType n, DvmType dep, DvmType type_of_run, DvmType *idxs) {

    int p[n];

    int where_ = where_dep(p, n, type_of_run, idxs, dep);

    bool cond = true;
    for (int i = 0; i < dep; i++)
        cond = cond && (p[i] == 1);
    if (cond) {
        // Nothing to do in that case
        return;
    }

    int depIdxs[n], inDepIdxs[n];
    DvmType tmpI[n], tmpH[n], tmpL[n];

    int depIdxsLen = 0, inDepIdxsLen = 0;
    for (int i = 0; i < n; ++i) {
        if (p[n - 1 - i] == 1)
            depIdxs[depIdxsLen++] = i;
        else
            inDepIdxs[inDepIdxsLen++] = i;
    }

    int count = 0;
    for (int i = 0; i < inDepIdxsLen; i++) {
        tmpI[count] = idx[inDepIdxs[i]];
        tmpL[count] = low[inDepIdxs[i]];
        tmpH[count] = high[inDepIdxs[i]];
        count++;
    }
    for (int i = 0; i < depIdxsLen; i++) {
        tmpI[count] = idx[depIdxs[i]];
        tmpL[count] = low[depIdxs[i]];
        tmpH[count] = high[depIdxs[i]];
        count++;
    }
    for (int i = 0; i < n; i++) {
        idx[i] = tmpI[i];
        low[i] = tmpL[i];
        high[i] = tmpH[i];
    }
}

__global__ void loop_adi_41_cuda_kernel(double *a, CudaIndexType a_hdr3, CudaIndexType a_hdr2, CudaIndexType a_hdr1, CudaIndexType num_elem_j, CudaIndexType num_elem_k, CudaIndexType base_i, CudaIndexType base_j, CudaIndexType base_k, CudaIndexType step_i, CudaIndexType step_j, CudaIndexType step_k, CudaIndexType idxs_0, CudaIndexType idxs_1, CudaIndexType idxs_2) {

    CudaIndexType id_x, id_y;
    CudaIndexType coords[3];

    int i;
    int j;
    int k;

    id_x = blockIdx.x * blockDim.x + threadIdx.x;
    id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (id_x < num_elem_j && id_y < num_elem_k) {
        coords[idxs_0] = base_i;
        coords[idxs_1] = base_j + id_x * step_j;
        coords[idxs_2] = base_k + id_y * step_k;
        i = coords[0];
        j = coords[1];
        k = coords[2];


            a[i * a_hdr3 + j * a_hdr2 + k * a_hdr1] = (a[(i - 1) * a_hdr3 + j * a_hdr2 + k * a_hdr1] + a[(i + 1) * a_hdr3 + j * a_hdr2 + k * a_hdr1]) / 2;

    }
}

extern "C" void loop_adi_41_cuda_across(DvmType *pLoopRef, DvmType a_hdr[], int dep_mask) {
    DvmType tmpVar;

    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);

    dvmh_loop_autotransform_C(loop_ref, a_hdr);
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);

    IndexType boundsLow[3], boundsHigh[3], loopSteps[3], idxs[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_41_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_41_cuda_kernel_regs, &threads, &stream, 0);


    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    change_filled_bounds(boundsLow, boundsHigh, loopSteps, 3, 1, dep_mask, idxs);


    int num_elem_x = (abs(boundsLow[1] - boundsHigh[1]) + 1) / abs(loopSteps[1]) + ((abs(boundsLow[1] - boundsHigh[1]) + 1) % abs(loopSteps[1]) != 0);
    int num_elem_y = (abs(boundsLow[0] - boundsHigh[0]) + 1) / abs(loopSteps[0]) + ((abs(boundsLow[0] - boundsHigh[0]) + 1) % abs(loopSteps[0]) != 0);

    int base_i = boundsLow[2];
    int base_j = boundsLow[1];
    int base_k = boundsLow[0];

    blocks.x = num_elem_x / threads.x + ((num_elem_x % threads.x != 0)?1:0);
    blocks.y = num_elem_y / threads.y + ((num_elem_y % threads.y != 0)?1:0);
    boundsHigh[2] = (abs(boundsHigh[2] - boundsLow[2]) + 1) / abs(loopSteps[2]);

    for (int tmpV = 0; tmpV < boundsHigh[2]; base_i += loopSteps[2], tmpV++) {
        loop_adi_41_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[3], a_devHdr[2], a_devHdr[1], num_elem_x, num_elem_y, base_i, base_j, base_k, loopSteps[2], loopSteps[1], loopSteps[0], idxs[0], idxs[1], idxs[2]);
    }
}

extern "C" void loop_adi_41_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    int dep_mask = dvmh_loop_get_dependency_mask_C(*pLoopRef);

    if (dep_mask == 1 || dep_mask == 2 || dep_mask == 4) {
        loop_adi_41_cuda_across(pLoopRef, a_hdr, dep_mask);
    }
}

__global__ void loop_adi_46_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType a_hdr3, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3) {
    /* Parameters */
    DvmhPermutatedArrayHelper3<double> a(a_base, a_hdr1, a_hdr2, a_hdr3);
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
            for (k = boundsLow_3; k <= boundsHigh_3; k++)
            {
                a[i][j][k] = (a[i][j - 1][k] + a[i][j + 1][k]) / 2;
            }
}

extern "C" void loop_adi_46_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, a_hdr);
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_46_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_46_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_46_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], a_devHdr[3], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], boundsLow[2], boundsHigh[2]);
        restBlocks -= blocks.x;
    }
}

__global__ void loop_adi_51_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType a_hdr3, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3, double eps, double eps_grid[]) {
    /* Parameters */
    DvmhPermutatedArrayHelper3<double> a(a_base, a_hdr1, a_hdr2, a_hdr3);
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    for (i = boundsLow_1; i <= boundsHigh_1; i++)
        for (j = boundsLow_2; j <= boundsHigh_2; j++)
            for (k = boundsLow_3; k <= boundsHigh_3; k++)
            {
              double tmp1 = (a[i][j][k - 1] + a[i][j][k + 1]) / 2;
              double tmp2 = fabs(a[i][j][k] - tmp1);
              eps = ((eps) > (tmp2) ? (eps) : (tmp2));
              a[i][j][k] = tmp1;
            }

    /* Write reduction values to global memory */
    eps_grid[0] = eps;
}

extern "C" void loop_adi_51_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, a_hdr);
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(1, 1, 1);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_51_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_51_cuda_kernel_regs, &threads, &stream, 0);
    threads = dim3(1, 1, 1);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);

    /* Reductions-related stuff */
    double eps;
    double *eps_grid;
    dvmh_loop_cuda_register_red_C(loop_ref, 1, (void **)&eps_grid, 0);
    dvmh_loop_red_init_C(loop_ref, 1, &eps, 0);
    dvmh_loop_cuda_red_prepare_C(loop_ref, 1, 1, 0);

    /* GPU execution */
    restBlocks = 1;
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_51_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], a_devHdr[3], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], boundsLow[2], boundsHigh[2], eps, eps_grid);
        restBlocks -= blocks.x;
    }

    dvmh_loop_cuda_red_finish_C(loop_ref, 1);
}

__global__ void loop_adi_92_cuda_kernel(double *a_base, CudaIndexType a_hdr1, CudaIndexType a_hdr2, CudaIndexType a_hdr3, CudaIndexType boundsLow_1, CudaIndexType boundsHigh_1, CudaIndexType boundsLow_2, CudaIndexType boundsHigh_2, CudaIndexType blocksS_2, CudaIndexType boundsLow_3, CudaIndexType boundsHigh_3, CudaIndexType blocksS_3, CudaIndexType blockOffset) {
    /* Parameters */
    DvmhPermutatedArrayHelper3<double> a(a_base, a_hdr1, a_hdr2, a_hdr3);
    /* Supplementary variables for loop handling */
    CudaIndexType restBlocks, curBlocks;
    /* User variables - loop index variables and other private variables */
    int i;
    int j;
    int k;

    restBlocks = blockIdx.x + blockOffset;
    curBlocks = restBlocks / blocksS_2;
    i = boundsLow_1 + (curBlocks * blockDim.z + threadIdx.z);
    if (i <= boundsHigh_1) {
        restBlocks = restBlocks - curBlocks * blocksS_2;
        curBlocks = restBlocks / blocksS_3;
        j = boundsLow_2 + (curBlocks * blockDim.y + threadIdx.y);
        if (j <= boundsHigh_2) {
            restBlocks = restBlocks - curBlocks * blocksS_3;
            curBlocks = restBlocks;
            k = boundsLow_3 + (curBlocks * blockDim.x + threadIdx.x);
            if (k <= boundsHigh_3)
            {
                if (k == 0 || k == 384 - 1 || j == 0 || j == 384 - 1 || i == 0 || i == 384 - 1)
                  a[i][j][k] = 10. * i / (384 - 1) + 10. * j / (384 - 1) + 10. * k / (384 - 1);
                else
                  a[i][j][k] = 0;
            }
        }
    }
}

extern "C" void loop_adi_92_cuda(DvmType *pLoopRef, DvmType a_hdr[]) {
    DvmType tmpVar;
    /* Loop reference and device number */
    DvmType loop_ref = *pLoopRef;
    DvmType device_num = dvmh_loop_get_device_num_C(loop_ref);
    /* Parameters */
    dvmh_loop_autotransform_C(loop_ref, a_hdr);
    double *a = (double *)dvmh_get_natural_base_C(device_num, a_hdr);
    DvmType a_devHdr[6];
    tmpVar = dvmh_fill_header_C(device_num, a, a_hdr, a_devHdr, 0);
    assert(tmpVar == 0 || tmpVar == 1);
    /* Supplementary variables for loop handling */
    IndexType boundsLow[3], boundsHigh[3], loopSteps[3];
    CudaIndexType blocksS[3];
    CudaIndexType restBlocks;
    dim3 blocks(1, 1, 1), threads(0, 0, 0);
    hipStream_t stream;

    /* Get CUDA configuration parameters */
    extern DvmType loop_adi_92_cuda_kernel_regs;
    dvmh_loop_cuda_get_config_C(loop_ref, 0, loop_adi_92_cuda_kernel_regs, &threads, &stream, 0);

    /* Calculate computation distribution parameters */
    dvmh_loop_fill_bounds_C(loop_ref, boundsLow, boundsHigh, loopSteps);
    blocksS[2] = ((boundsHigh[2] - boundsLow[2]) / loopSteps[2] + 1 + (threads.x - 1)) / threads.x;
    blocksS[1] = blocksS[2] * (((boundsHigh[1] - boundsLow[1]) / loopSteps[1] + 1 + (threads.y - 1)) / threads.y);
    blocksS[0] = blocksS[1] * (((boundsHigh[0] - boundsLow[0]) / loopSteps[0] + 1 + (threads.z - 1)) / threads.z);

    /* GPU execution */
    restBlocks = blocksS[0];
    while (restBlocks > 0) {
        blocks.x = (restBlocks <= 65535 ? restBlocks : (restBlocks <= 65535 * 2 ? restBlocks / 2 : 65535));
        loop_adi_92_cuda_kernel<<<blocks, threads, 0, stream>>>(a, a_devHdr[1], a_devHdr[2], a_devHdr[3], boundsLow[0], boundsHigh[0], boundsLow[1], boundsHigh[1], blocksS[1], boundsLow[2], boundsHigh[2], blocksS[2], blocksS[0] - restBlocks);
        restBlocks -= blocks.x;
    }
}

